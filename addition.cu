// nvcc -o matrixmul -arch=sm_86 -Xptxas -O3,-v matrixmul.cu
// cuobjdump vec_add -sass
// nvcc -o matrixmul -arch=sm_86 -g -G matrixmul.cu


#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cassert>
#include <string>
#include "hip/hip_runtime_api.h"

using namespace std;

#define blockSize 128

#define CUDACHECK(cmd) do {                         \
  hipError_t err = cmd;                            \
  if (err != hipSuccess) {                         \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(err)); \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


// This function prints basic information about the device
void printInfo()
{
  int nDevices;
  hipGetDeviceCount(&nDevices);
  cout << string(50, '-') << endl;
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    cout << "Device Number: " << i << endl;
    cout << "  Device name: " << prop.name << endl;
    cout << "  SM count: " << prop.multiProcessorCount << endl;
    cout << "  Max Blocks per SM: " << prop.maxBlocksPerMultiProcessor << endl;
    cout << "  Max Threads per Block: " << prop.maxThreadsPerBlock << endl;
    cout << "  Max Threads per SM: " << prop.maxThreadsPerMultiProcessor << endl;
    assert(blockSize <= prop.maxThreadsPerBlock);
  }
  cout << string(50, '-') << endl;
}

// This function reads an input file and copy the data to the host vector
void readInputFiles(vector<float> &v)
{
  ifstream f("inp1.txt");
  float a;
  while (f >> a)
    v.push_back(a);
}

// This helper function prints the final results on the screen
void print(vector<float> v)
{
  for (auto x:v)
    cout << x << " ";
  cout << "\n";
}

// This function allocates the array on device memory and copy inout data from host to device
// Note that std::vector::data() returns a pointer to the first location of vector, a.k.a hv[0]
void allocateOnDevice(float *&dv, vector<float> &hv, int n)
{
  CUDACHECK( hipMalloc((void**)&dv, n * sizeof(float)) );
  CUDACHECK( hipMemcpy(dv, hv.data(), n * sizeof(float), hipMemcpyHostToDevice) );
}

// This function transfers the result from device to host
void transferResults(float *&dv, vector<float> &hv, int n)
{
  hv.resize(n);
  CUDACHECK( hipMemcpy(hv.data(), dv, n * sizeof(float), hipMemcpyDeviceToHost) );
  print(hv);
}

// This function frees device memory
void freeMemory(float *&dv)
{
  CUDACHECK(hipFree(dv));
}


// This function returns the SM number
__device__ uint get_smid(void) {
     uint ret;
     asm("mov.u32 %0, %smid;" : "=r"(ret) );
     return ret;
}


// Kernel inplementation
__global__ void simpleAdd(float *v, int n)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    printf("SM(%d) Block(%d,%d,%d) Thread(%d,%d,%d) -> %.2f + 1\n", get_smid(), blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z, v[i]);
    v[i] = v[i] + 1;
  }
}

    
int main()
{
  printInfo();
  vector<float> hostVector;
  readInputFiles(hostVector);
  cout << "Read " << hostVector.size() << " elements from inp1.txt\n";

  int n = hostVector.size();
  float *deviceVector;
  allocateOnDevice(deviceVector, hostVector, n);

  int numBlocks = (n + blockSize - 1) / blockSize;
  
  hipEvent_t start, stop;
  float elapsedTime;
  hipEventCreate(&start); hipEventRecord(start,0);

  simpleAdd<<<numBlocks,blockSize>>>(deviceVector, n);

  hipEventCreate(&stop);  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start,stop);

  cout << "addABC elapsed time : " << elapsedTime << " ms\n";
  hipDeviceSynchronize();

  transferResults(deviceVector, hostVector, n);
  freeMemory(deviceVector);

  cout << "Done\n";
  return 0;
}

